
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <driver_functions.h>
#include <hip/driver_types.h>
#include <optional>

#define gpuErrchk(ans)                                                         \
  {                                                                            \
    gpuAssert((ans), __FILE__, __LINE__);                                      \
  }

#define gpuErrchkPassthrough(ans)                                              \
  {                                                                            \
    gpuAssert((ans), __FILE__, __LINE__, false);                               \
  }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

/**
 * Returns a resulting uchar mask
 * Bounds are inclusive. Not specifying a bound assumes valid range is (0, 255]
 */
__global__ void JoinUCharMasks(hipPitchedPtr masks, hipPitchedPtr outMask,
                               size_t bandDimX, size_t bandDimY, size_t nMasks,
                               unsigned char boundMin = 1,
                               unsigned char boundMax = 255) {
  int r = blockDim.x * blockIdx.x + threadIdx.x;
  int c = blockDim.y * blockIdx.y + threadIdx.y;

  if (c > bandDimX || r > bandDimY) {
    return;
  }

  size_t maskPitch = masks.pitch * bandDimY;
  size_t maskRowPitch = masks.pitch;

  size_t outRowPitch = outMask.pitch;

  for (size_t maskNum = 0; maskNum < nMasks; maskNum++) {
    const unsigned char *mask =
        (unsigned char *)masks.ptr + maskNum * maskPitch;

    const unsigned char *maskRow = mask + maskRowPitch * r;
    unsigned char *outRow = (unsigned char *)outMask.ptr + outRowPitch * r;

    char maskValue = maskRow[c];

    if (maskValue < boundMin || maskValue > boundMax) {
      outRow[c] = 0;
    }
  }
}

void joinDetfooMasks(unsigned char *masks, unsigned char *outMask,
                     size_t bandDimX, size_t bandDimY, size_t nMasks) {
  hipPitchedPtr d_masksPtr;
  hipExtent masksExtent =
      make_hipExtent(bandDimX * sizeof(unsigned char), bandDimY, nMasks);
  gpuErrchk(hipMalloc3D(&d_masksPtr, masksExtent));

  hipMemcpy3DParms masksCopyParams = {};
  masksCopyParams.kind = hipMemcpyKind::hipMemcpyHostToDevice;
  masksCopyParams.extent = masksExtent;
  masksCopyParams.dstPtr = d_masksPtr;
  masksCopyParams.srcPtr = make_hipPitchedPtr(
      (void *)masks, sizeof(unsigned char) * bandDimX, bandDimX, bandDimY);
  gpuErrchk(hipMemcpy3D(&masksCopyParams));

  hipPitchedPtr d_outPtr;
  d_outPtr.xsize = sizeof(unsigned char) * bandDimX;
  d_outPtr.ysize = bandDimY;
  gpuErrchk(hipMallocPitch(&d_outPtr.ptr, &d_outPtr.pitch, d_outPtr.xsize,
                            d_outPtr.ysize));
  gpuErrchk(hipMemset2D(d_outPtr.ptr, d_outPtr.pitch, 1,
                         bandDimX * sizeof(unsigned char), bandDimY));

  const int threadDim = 16;
  dim3 threadsPerBlock(threadDim, threadDim);
  dim3 blocksPerGrid(bandDimX / threadDim, bandDimY / threadDim);

  if (bandDimX % threadDim) {
    blocksPerGrid.x++;
  }

  if (bandDimY % threadDim) {
    blocksPerGrid.y++;
  }

  JoinUCharMasks<<<blocksPerGrid, threadsPerBlock>>>(
      d_masksPtr, d_outPtr, bandDimX, bandDimY, nMasks);
  gpuErrchk(hipPeekAtLastError());

  gpuErrchkPassthrough(hipFree(d_masksPtr.ptr));

  gpuErrchk(hipMemcpy2D((void *)outMask, sizeof(unsigned char) * bandDimX,
                         d_outPtr.ptr, d_outPtr.pitch,
                         bandDimX * sizeof(unsigned char), bandDimY,
                         hipMemcpyKind::hipMemcpyDeviceToHost));
  gpuErrchkPassthrough(hipFree(d_outPtr.ptr));
}

void joinMasks(unsigned char *detfooMasks, size_t nDetfooMasks,
               unsigned char *cldMask, unsigned char maxCldPercentage,
               unsigned char *snwMask, unsigned char maxSnwPercentage,
               unsigned char *outMask, size_t bandDimX, size_t bandDimY) {

  // join detfoo masks
  hipPitchedPtr d_detfooMasksPtr;
  hipExtent masksExtent =
      make_hipExtent(bandDimX * sizeof(unsigned char), bandDimY, nDetfooMasks);
  gpuErrchk(hipMalloc3D(&d_detfooMasksPtr, masksExtent));

  hipMemcpy3DParms masksCopyParams = {};
  masksCopyParams.kind = hipMemcpyKind::hipMemcpyHostToDevice;
  masksCopyParams.extent = masksExtent;
  masksCopyParams.dstPtr = d_detfooMasksPtr;
  masksCopyParams.srcPtr =
      make_hipPitchedPtr((void *)detfooMasks, sizeof(unsigned char) * bandDimX,
                          bandDimX, bandDimY);
  gpuErrchk(hipMemcpy3D(&masksCopyParams));

  hipPitchedPtr d_outPtr;
  d_outPtr.xsize = sizeof(unsigned char) * bandDimX;
  d_outPtr.ysize = bandDimY;
  gpuErrchk(hipMallocPitch(&d_outPtr.ptr, &d_outPtr.pitch, d_outPtr.xsize,
                            d_outPtr.ysize));
  gpuErrchk(hipMemset2D(d_outPtr.ptr, d_outPtr.pitch, 1,
                         bandDimX * sizeof(unsigned char), bandDimY));

  const int threadDim = 16;
  dim3 threadsPerBlock(threadDim, threadDim);
  dim3 blocksPerGrid(bandDimX / threadDim, bandDimY / threadDim);

  if (bandDimX % threadDim) {
    blocksPerGrid.x++;
  }

  if (bandDimY % threadDim) {
    blocksPerGrid.y++;
  }

  JoinUCharMasks<<<blocksPerGrid, threadsPerBlock>>>(
      d_detfooMasksPtr, d_outPtr, bandDimX, bandDimY, nDetfooMasks);
  gpuErrchk(hipPeekAtLastError());

  gpuErrchkPassthrough(hipFree(d_detfooMasksPtr.ptr));

  // join cld mask
  hipPitchedPtr d_cldMaskPtr = {};
  d_cldMaskPtr.xsize = bandDimX;
  d_cldMaskPtr.ysize = bandDimY;
  gpuErrchk(hipMallocPitch(&d_cldMaskPtr.ptr, &d_cldMaskPtr.pitch,
                            d_cldMaskPtr.xsize * sizeof(unsigned char),
                            d_cldMaskPtr.ysize));
  gpuErrchk(hipMemcpy2D(d_cldMaskPtr.ptr, d_cldMaskPtr.pitch, (void *)cldMask,
                         bandDimX * sizeof(unsigned char),
                         bandDimX * sizeof(unsigned char), bandDimY,
                         hipMemcpyHostToDevice));
  JoinUCharMasks<<<blocksPerGrid, threadsPerBlock>>>(
      d_cldMaskPtr, d_outPtr, bandDimX, bandDimY, 1, 0, maxCldPercentage);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchkPassthrough(hipFree(d_cldMaskPtr.ptr));

  // join snw mask
  hipPitchedPtr d_snwMaskPtr = {};
  d_snwMaskPtr.xsize = bandDimX;
  d_snwMaskPtr.ysize = bandDimY;
  gpuErrchk(hipMallocPitch(&d_snwMaskPtr.ptr, &d_snwMaskPtr.pitch,
                            d_snwMaskPtr.xsize * sizeof(unsigned char),
                            d_snwMaskPtr.ysize));
  gpuErrchk(hipMemcpy2D(d_snwMaskPtr.ptr, d_snwMaskPtr.pitch, (void *)snwMask,
                         bandDimX * sizeof(unsigned char),
                         bandDimX * sizeof(unsigned char), bandDimY,
                         hipMemcpyHostToDevice));
  JoinUCharMasks<<<blocksPerGrid, threadsPerBlock>>>(
      d_snwMaskPtr, d_outPtr, bandDimX, bandDimY, 1, 0, maxSnwPercentage);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchkPassthrough(hipFree(d_snwMaskPtr.ptr));

  // copy result to host
  gpuErrchk(hipMemcpy2D((void *)outMask, sizeof(unsigned char) * bandDimX,
                         d_outPtr.ptr, d_outPtr.pitch,
                         bandDimX * sizeof(unsigned char), bandDimY,
                         hipMemcpyKind::hipMemcpyDeviceToHost));
  gpuErrchkPassthrough(hipFree(d_outPtr.ptr));
}
