
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdio>
#include <iostream>

#define gpuErrchk(ans)                                                         \
  {                                                                            \
    gpuAssert((ans), __FILE__, __LINE__);                                      \
  }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__global__ void BuildRowNonzeroSums(hipPitchedPtr bands, int *out,
                                    size_t out_pitch, int bandDimX,
                                    int bandDimY, int nBands, int sampleSize) {
  int r = blockDim.x * blockIdx.x + threadIdx.x;
  int c = blockDim.y * blockIdx.y + threadIdx.y;

  if (c > bandDimX - sampleSize || r > bandDimY - sampleSize) {
    return;
  }

  int pitch = bands.pitch;
  int slicePitch = bands.pitch * bandDimY;

  int *outRow = (int *)((char *)out + pitch * r);
  outRow[c] = 0;

  for (int band = 0; band < nBands; band++) {
    char *bandSlice = (char *)bands.ptr + band * slicePitch;
    float *row = (float *)(bandSlice + r * pitch);

    for (int k = c; k < c + sampleSize; c++) {
      if (row[k] != 0.0f) {
        outRow[c] = 0;
      }
    }
  }

  // for (int band = 0; band < nBands; band++) {
  //   for (int k = i; k < i + sampleSize; k++) {
  //     out[i][j] = 0;
  //     for (int l = j; l < j + sampleSize; l++) {
  //       out[k][l]++;
  //     }
  //   }
  // }
}

__global__ void BuildSampleMap(int *rowSums, size_t rowSums_pitch, int *out,
                               size_t out_pitch, int bandDimX, int bandDimY,
                               int sampleSize) {
  int r = blockDim.x * blockIdx.x + threadIdx.x;
  int c = blockDim.y * blockIdx.y + threadIdx.y;

  if (c > bandDimX - sampleSize || r > bandDimY - sampleSize) {
    return;
  }

  int *outRow = (int *)((char *)out + out_pitch * r);
  outRow[c] = 0;

  for (int k = r; k < r + sampleSize; r++) {
    int *rowSumsRow = (int *)((char *)rowSums + rowSums_pitch * k);

    outRow[c] += rowSumsRow[c];
  }
}

void buildSampleMap(float ***bands, int **out, int bandDimX, int bandDimY,
                    int nBands, int sampleSize) {
  hipError_t setErr = hipSetDevice(0);

  if (setErr == hipErrorInvalidDevice) {
    std::cout << "invalid device!" << std::endl;
  } else if (setErr == cudaErrorDevicesUnavailable) {
    std::cout << "devices unavailable!" << std::endl;
  }

  hipPitchedPtr d_bandsPtr;
  hipExtent extent =
      (make_hipExtent(bandDimX * sizeof(float), bandDimY, nBands));
  std::cout << "allocating: " << extent.width * extent.depth * extent.height
            << std::endl;

  size_t total, free, temp, used;
  setErr = hipMemGetInfo(&temp, &total); // get memory information

  std::cout << "setErr: " << setErr << " " << hipGetErrorName(setErr)
            << std::endl;

  printf("Total mem: %lu \t free mem before malloc: %lu\n", total,
         temp); // output

  gpuErrchk(hipMalloc3D(&d_bandsPtr, extent));

  hipMemcpy3DParms copyParams = {};
  copyParams.extent = extent;
  copyParams.kind = hipMemcpyKind::hipMemcpyHostToDevice;
  copyParams.dstPtr = d_bandsPtr;
  copyParams.srcPtr =
      make_hipPitchedPtr(bands, sizeof(float) * bandDimX, bandDimX, bandDimY);
  gpuErrchk(hipMemcpy3D(&copyParams));

  int *d_rowSums;
  size_t d_rowSums_pitch;
  gpuErrchk(hipMallocPitch(&d_rowSums, &d_rowSums_pitch,
                            bandDimX * (sizeof(int)), bandDimY));

  const int threadDim = 16;
  dim3 threadsPerBlock(threadDim, threadDim);
  dim3 blocksPerGrid(bandDimX / threadDim + (threadDim - bandDimX % threadDim),
                     bandDimY / threadDim + (threadDim - bandDimY % threadDim));
  BuildRowNonzeroSums<<<blocksPerGrid, threadsPerBlock>>>(
      d_bandsPtr, d_rowSums, d_rowSums_pitch, bandDimX, bandDimY, nBands,
      sampleSize);
  gpuErrchk(hipPeekAtLastError());

  int *d_out;
  size_t d_out_pitch;
  gpuErrchk(hipMallocPitch(&d_out, &d_out_pitch, bandDimX * (sizeof(int)),
                            bandDimY));
  BuildSampleMap<<<blocksPerGrid, threadsPerBlock>>>(
      d_rowSums, d_rowSums_pitch, d_out, d_out_pitch, bandDimX, bandDimY,
      sampleSize);
  gpuErrchk(hipPeekAtLastError());
}
