#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <iostream>

#include "cuda/mapgen.h"
#include "cuda/maskJoin.cuh"

// #define gpuErrchk(ans)                                                         \
//   {                                                                            \
//     gpuAssert((ans), __FILE__, __LINE__);                                      \
//   }
// static inline void gpuAssert(hipError_t code, const char *file, int line,
//                              bool abort = true) {
//   if (code != hipSuccess) {
//     fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
//             line);
//     if (abort)
//       exit(code);
//   }
// }

__global__ void BuildRowNonzeroSums(hipPitchedPtr bands, int *out,
                                    size_t out_pitch, size_t bandDimX,
                                    size_t bandDimY, int nBands,
                                    size_t sampleSize) {
  int r = blockDim.x * blockIdx.x + threadIdx.x;
  int c = blockDim.y * blockIdx.y + threadIdx.y;

  int pitch = bands.pitch;
  int slicePitch = bands.pitch * bandDimY;

  int *outRow = (int *)((char *)out + out_pitch * r);

  if ((c > bandDimX - sampleSize) || (r > bandDimY - sampleSize)) {
    if (c < bandDimX && r < bandDimY) {
      outRow[c] = 0;
    }
    return;
  }
  outRow[c] = 0;

  for (int band = 0; band < nBands; band++) {
    unsigned char *bandSlice = (unsigned char *)bands.ptr + band * slicePitch;
    unsigned char *row = (unsigned char *)(bandSlice + r * bands.pitch);

    for (int k = c; k < c + sampleSize; k++) {
      if (row[k] != 0) {
        outRow[c] = outRow[c] + 1;
      }
    }
  }

  // for (int band = 0; band < nBands; band++) {
  //   for (int k = i; k < i + sampleSize; k++) {
  //     out[i][j] = 0;
  //     for (int l = j; l < j + sampleSize; l++) {
  //       out[k][l]++;
  //     }
  //   }
  // }
}

__global__ void BuildSampleMap(int *rowSums, size_t rowSums_pitch,
                               unsigned char *out, size_t out_pitch,
                               int bandDimX, int bandDimY, int sampleSize,
                               float minNonzeroPercentage) {
  int r = blockDim.x * blockIdx.x + threadIdx.x;
  int c = blockDim.y * blockIdx.y + threadIdx.y;

  unsigned char *outRow = (unsigned char *)(out + out_pitch * r);
  if ((c > (bandDimX - sampleSize)) || (r > (bandDimY - sampleSize))) {
    if (c < bandDimX && r < bandDimY) {
      outRow[c] = 0;
    }
    return;
  }

  outRow[c] = 0;

  size_t total = 0;

  for (int k = r; k < r + sampleSize; k++) {
    int *rowSumsRow = (int *)((char *)rowSums + rowSums_pitch * k);

    total += rowSumsRow[c];
  }

  float percentage = (float)total / sampleSize / sampleSize;

  if (percentage > minNonzeroPercentage) {
    outRow[c] = 1;
  }
}

// void buildSampleMap(float ***bands, int **out, int bandDimX, int bandDimY,
//                     int nBands, int sampleSize) {
//   hipError_t setErr = hipSetDevice(0);
//
//   if (setErr == hipErrorInvalidDevice) {
//     std::cout << "invalid device!" << std::endl;
//   } else if (setErr == cudaErrorDevicesUnavailable) {
//     std::cout << "devices unavailable!" << std::endl;
//   }
//
//   size_t total, free, temp, used;
//   setErr = hipMemGetInfo(&temp, &total); // get memory information
//
//   // std::cout << "setErr: " << setErr << " " << hipGetErrorName(setErr)
//   //           << std::endl;
//   //
//   // printf("Total mem: %lu \t free mem before malloc: %lu\n", total,
//   //        temp); // output
//
//   hipPitchedPtr d_bandsPtr;
//   hipExtent extent =
//       (make_hipExtent(bandDimX * sizeof(float), bandDimY, nBands));
//   std::cout << "allocating: " << extent.width * extent.depth * extent.height
//             << std::endl;
//
//   gpuErrchk(hipMalloc3D(&d_bandsPtr, extent));
//
//   hipMemcpy3DParms copyParams = {};
//   copyParams.extent = extent;
//   copyParams.kind = hipMemcpyKind::hipMemcpyHostToDevice;
//   copyParams.dstPtr = d_bandsPtr;
//   copyParams.srcPtr =
//       make_hipPitchedPtr(bands, sizeof(float) * bandDimX, bandDimX,
//       bandDimY);
//   gpuErrchk(hipMemcpy3D(&copyParams));
//
//   int *d_rowSums;
//   size_t d_rowSums_pitch;
//   gpuErrchk(hipMallocPitch(&d_rowSums, &d_rowSums_pitch,
//                             bandDimX * (sizeof(int)), bandDimY));
//
//   const int threadDim = 16;
//   dim3 threadsPerBlock(threadDim, threadDim);
//   dim3 blocksPerGrid(bandDimX / threadDim + (threadDim - bandDimX %
//   threadDim),
//                      bandDimY / threadDim + (threadDim - bandDimY %
//                      threadDim));
//   BuildRowNonzeroSums<<<blocksPerGrid, threadsPerBlock>>>(
//       d_bandsPtr, d_rowSums, d_rowSums_pitch, bandDimX, bandDimY, nBands,
//       sampleSize);
//   gpuErrchk(hipPeekAtLastError());
//
//   int *d_out;
//   size_t d_out_pitch;
//   gpuErrchk(hipMallocPitch(&d_out, &d_out_pitch, bandDimX * (sizeof(int)),
//                             bandDimY));
//   BuildSampleMap<<<blocksPerGrid, threadsPerBlock>>>(
//       d_rowSums, d_rowSums_pitch, d_out, d_out_pitch, bandDimX, bandDimY,
//       sampleSize);
//   gpuErrchk(hipPeekAtLastError());
// }

namespace sats::cudaproc {
void generateSampleMap(unsigned char *detfooMasks, size_t nDetfooMasks,
                       unsigned char *cldMask, unsigned char maxCldPercentage,
                       unsigned char *snwMask, unsigned char maxSnwPercentage,
                       unsigned char *outMask, size_t bandDimX, size_t bandDimY,
                       size_t sampleSize, float minNonzeroPercentage) {

  // join detfoo masks
  hipPitchedPtr d_detfooMasksPtr;
  hipExtent masksExtent =
      make_hipExtent(bandDimX * sizeof(unsigned char), bandDimY, nDetfooMasks);
  gpuErrchk(hipMalloc3D(&d_detfooMasksPtr, masksExtent));

  hipMemcpy3DParms masksCopyParams = {};
  masksCopyParams.kind = hipMemcpyKind::hipMemcpyHostToDevice;
  masksCopyParams.extent = masksExtent;
  masksCopyParams.dstPtr = d_detfooMasksPtr;
  masksCopyParams.srcPtr =
      make_hipPitchedPtr((void *)detfooMasks, sizeof(unsigned char) * bandDimX,
                          bandDimX, bandDimY);
  gpuErrchk(hipMemcpy3D(&masksCopyParams));

  hipPitchedPtr d_outPtr;
  d_outPtr.xsize = sizeof(unsigned char) * bandDimX;
  d_outPtr.ysize = bandDimY;
  gpuErrchk(hipMallocPitch(&d_outPtr.ptr, &d_outPtr.pitch, d_outPtr.xsize,
                            d_outPtr.ysize));
  gpuErrchk(hipMemset2D(d_outPtr.ptr, d_outPtr.pitch, 1,
                         bandDimX * sizeof(unsigned char), bandDimY));

  const int threadDim = 16;
  dim3 threadsPerBlock(threadDim, threadDim);
  dim3 blocksPerGrid(bandDimX / threadDim, bandDimY / threadDim);

  if (bandDimX % threadDim) {
    blocksPerGrid.x++;
  }

  if (bandDimY % threadDim) {
    blocksPerGrid.y++;
  }

  JoinUCharMasks<<<blocksPerGrid, threadsPerBlock>>>(
      d_detfooMasksPtr, d_outPtr, bandDimX, bandDimY, nDetfooMasks);
  hipStreamSynchronize(0);
  gpuErrchk(hipPeekAtLastError());

  gpuErrchkPassthrough(hipFree(d_detfooMasksPtr.ptr));

  // join cld mask
  hipPitchedPtr d_cldMaskPtr = {};
  d_cldMaskPtr.xsize = bandDimX;
  d_cldMaskPtr.ysize = bandDimY;
  gpuErrchk(hipMallocPitch(&d_cldMaskPtr.ptr, &d_cldMaskPtr.pitch,
                            d_cldMaskPtr.xsize * sizeof(unsigned char),
                            d_cldMaskPtr.ysize));
  gpuErrchk(hipMemcpy2D(d_cldMaskPtr.ptr, d_cldMaskPtr.pitch, (void *)cldMask,
                         bandDimX * sizeof(unsigned char),
                         bandDimX * sizeof(unsigned char), bandDimY,
                         hipMemcpyHostToDevice));
  JoinUCharMasks<<<blocksPerGrid, threadsPerBlock>>>(
      d_cldMaskPtr, d_outPtr, bandDimX, bandDimY, 1, 0, maxCldPercentage);
  hipStreamSynchronize(0);
  gpuErrchk(hipPeekAtLastError());
  // join snw mask
  hipPitchedPtr d_snwMaskPtr = {};
  d_snwMaskPtr.xsize = bandDimX;
  d_snwMaskPtr.ysize = bandDimY;
  gpuErrchk(hipMallocPitch(&d_snwMaskPtr.ptr, &d_snwMaskPtr.pitch,
                            d_snwMaskPtr.xsize * sizeof(unsigned char),
                            d_snwMaskPtr.ysize));
  gpuErrchk(hipMemcpy2D(d_snwMaskPtr.ptr, d_snwMaskPtr.pitch, (void *)snwMask,
                         bandDimX * sizeof(unsigned char),
                         bandDimX * sizeof(unsigned char), bandDimY,
                         hipMemcpyHostToDevice));
  JoinUCharMasks<<<blocksPerGrid, threadsPerBlock>>>(
      d_snwMaskPtr, d_outPtr, bandDimX, bandDimY, 1, 0, maxSnwPercentage);
  hipStreamSynchronize(0);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchkPassthrough(hipFree(d_snwMaskPtr.ptr));

  // std::cout << "finished b" << std::endl;

  // const int threadDim = 16;
  // dim3 threadsPerBlock(threadDim, threadDim);
  // dim3 blocksPerGrid(bandDimX / threadDim + (threadDim - bandDimX %
  // threadDim),
  //                    bandDimY / threadDim + (threadDim - bandDimY %
  //                    threadDim));
  int *d_rowSums;
  size_t d_rowSums_pitch;
  gpuErrchk(hipMallocPitch(&d_rowSums, &d_rowSums_pitch,
                            bandDimX * (sizeof(int)), bandDimY));

  BuildRowNonzeroSums<<<blocksPerGrid, threadsPerBlock>>>(
      d_outPtr, d_rowSums, d_rowSums_pitch, bandDimX, bandDimY, 1, sampleSize);
  hipStreamSynchronize(0);
  gpuErrchk(hipPeekAtLastError());

  // std::cout << "finished a" << std::endl;

  BuildSampleMap<<<blocksPerGrid, threadsPerBlock>>>(
      d_rowSums, d_rowSums_pitch, (unsigned char *)d_outPtr.ptr, d_outPtr.pitch,
      bandDimX, bandDimY, sampleSize, minNonzeroPercentage);
  gpuErrchk(hipPeekAtLastError());

  hipStreamSynchronize(0);
  gpuErrchk(hipPeekAtLastError());
  // std::cout << "finished" << std::endl;

  // copy result to host
  gpuErrchk(hipMemcpy2D((void *)outMask, sizeof(unsigned char) * bandDimX,
                         d_outPtr.ptr, d_outPtr.pitch,
                         bandDimX * sizeof(unsigned char), bandDimY,
                         hipMemcpyKind::hipMemcpyDeviceToHost));

  gpuErrchkPassthrough(hipFree(d_rowSums));
  gpuErrchkPassthrough(hipFree(d_outPtr.ptr));
}
} // namespace sats::cudaproc
